/*
Hello world of wave propagation in CUDA. FDTD acoustic wave propagation in homogeneous medium. Second order in space and time 
*/

#include "stdio.h"
#include "math.h"
#include "stdlib.h"
#include "string.h"

/*
Add this to c_cpp_properties.json if linting isn't working for cuda libraries
"includePath": [
                "/usr/local/cuda-10.0/targets/x86_64-linux/include",
                "${workspaceFolder}/**"
            ],
*/

#include "hip/hip_runtime.h"


// Check error codes for CUDA functions
#define CHECK(call)                                                \
    {                                                              \
        hipError_t error = call;                                  \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
        }                                                          \
    }

#define PI 3.14159265359

#define HALO 4
#define HALO2 8

#define a0  -2.8472222f
#define a1   1.6000000f
#define a2  -0.2000000f
#define a3   0.0253968f
#define a4  -0.0017857f

#define BDIMX 32
#define BDIMY 32

#define SDIMX BDIMX + HALO2
#define SDIMY BDIMY + HALO2

// Allocate the constant device memory
__constant__ float c_coef[5]; /* coefficients for 8th order fd */
__constant__ int c_isrc;      /* source location, ox */
__constant__ int c_jsrc;      /* source location, oz */
__constant__ int c_nx;        /* x dim */
__constant__ int c_ny;        /* y dim */
__constant__ int c_nt;        /* time steps */
__constant__ int c_dt2dx2;    /* dt2 / dx2 for fd*/

// Save snapshot as a binary
void saveSnapshotIstep(int istep, float *data, int nx, int ny, const char *tag)
{
    float *iwave = (float *)malloc(nx * ny * sizeof(float));

    unsigned int isize = nx * ny;
    CHECK(hipMemcpy(iwave, data, isize * sizeof(float), hipMemcpyDeviceToHost));

    char fname[32];
    sprintf(fname, "snap/snap_%s_%i_%i_%i", tag, istep, ny, nx);

    FILE *fp_snap = fopen(fname, "w");

    fwrite(iwave, sizeof(float), nx * ny, fp_snap);
    printf("%s: nx = %i ny = %i istep = %i tag = %s\n", fname, nx, ny, istep, tag);
    fflush(stdout);
    fclose(fp_snap);

    free(iwave);
    return;
}

// Add source wavelet
__global__ void kernel_add_wavelet(float *d_u, float *d_wavelet, int it)
{
    unsigned int gx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int gy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = gy * c_nx + gx;

    if ((gx == c_isrc) && (gy == c_jsrc))
    {
        d_u[idx] += d_wavelet[it];
        // d_u[idx] = 1.0;

        // printf("\t%i\t%f\t%f\n",it, d_u[idx], d_wavelet[it]);
    }
}

__device__ void set_halo(float *global, float shared[][SDIMX], int tx, int ty, int sx, int sy, int gx, int gy, int nx, int ny)
{
    // fill inner smem
    shared[sy][sx] = global[gy * nx + gx];

    // LEFT
    if (tx < HALO)
    {
        if (gx < HALO)
        {
            // if global left
            shared[sy][sx - HALO] = 0.0;
        }
        else
        {
            // if block left
            shared[sy][sx - HALO] = global[gy * nx + gx - HALO];
        }
    }
    // RIGHT
    if ((tx >= (BDIMX - HALO)) || ((gx + HALO) >= nx))
    {
        if ((gx + HALO) >= nx)
        {
            // if global right
            shared[sy][sx + HALO] = 0.0;
        }
        else
        {
            // if block right
            shared[sy][sx + HALO] = global[gy * nx + gx + HALO];
        }
    }

    // BOTTOM
    if (ty < HALO)
    {
        if (gy < HALO)
        {
            // if global bottom
            shared[sy - HALO][sx] = 0.0;
        }
        else
        {
            // if block bottom
            shared[sy - HALO][sx] = global[(gy - HALO) * nx + gx];
        }
    }

    // TOP
    if ((ty >= (BDIMY - HALO)) || ((gy + HALO) >= ny))
    {
        if ((gy + HALO) >= ny)
        {
            // if global top
            shared[sy + HALO][sx] = 0.0;
        }
        else
        {
            // if block top
            shared[sy + HALO][sx] = global[(gy + HALO) * nx + gx];
        }
    }
}

// FD kernel
__global__ void kernel_2dfd(float *d_u1, float *d_u2, float *d_vp, float dt2dx2)
{
    // save model dims in registers as they are much faster
    const int nx = c_nx;
    const int ny = c_ny;

    // fd coef dt2 / dx2
    // const float dt2dx2 = c_dt2dx2;

    // thread addres (ty, tx) in a block
    const unsigned int tx = threadIdx.x;
    const unsigned int ty = threadIdx.y;

    // thread addres (sy, sx) in shared memory
    const unsigned int sx = threadIdx.x + HALO;
    const unsigned int sy = threadIdx.y + HALO;

    // thread address (gy, gx) in global memory
    const unsigned int gx = blockIdx.x * blockDim.x + tx;
    const unsigned int gy = blockIdx.y * blockDim.y + ty;

    // global linear index
    const unsigned int idx = gy * nx + gx;

    // Allocate shared memory(smem)
    __shared__ float s_u1[SDIMY][SDIMX];
    __shared__ float s_u2[SDIMY][SDIMX];
    __shared__ float s_vp[SDIMY][SDIMX];
    // __shared__ float tmp[SDIMY][SDIMX];


    // if thread points into the model
    if ((gx < nx) && (gy < ny))
    {
        // Copy regions from gmem into smem
        //       gmem, smem,  block, shared, global, dims
        set_halo(d_u1, s_u1, tx, ty, sx, sy, gx, gy, nx, ny);
        set_halo(d_u2, s_u2, tx, ty, sx, sy, gx, gy, nx, ny);
        set_halo(d_vp, s_vp, tx, ty, sx, sy, gx, gy, nx, ny);
        __syncthreads();

        float du2_xx = c_coef[0] * s_u2[sy][sx];
        float du2_yy = c_coef[0] * s_u2[sy][sx];

#pragma unroll
        for (int d = 1; d <= 4; d++)
        {
            du2_xx += c_coef[d] * (s_u2[sy][sx - d] + s_u2[sy][sx + d]);
            du2_yy += c_coef[d] * (s_u2[sy - d][sx] + s_u2[sy + d][sx]);
        }
        // if ((gx == c_isrc-1) && (gy == c_jsrc-1))
        // {
        //     // printf("%e %e %e %e %e %e\n", d_u3[idx], s_u2[sy][sx],s_u1[sy][sx],du2_xx, du2_yy,dt2dx2);
        //     printf("\t%i %i %i %i %e %e\n", c_isrc, c_jsrc, c_nx, c_ny, c_dt2dx2, dt2dx2); 
        //     // printf("\t\t%f\t%f\t%f\t%f\t%f\n", c_coef[0], c_coef[1], c_coef[2], c_coef[3], c_coef[4]);
        // }

        // d_u3[idx] = 2.0 * s_u2[sy][sx] - s_u1[sy][sx] + s_vp[sy][sx] * s_vp[sy][sx] * (du2_xx + du2_yy) * dt2dx2;
        d_u1[idx] = 2.0 * s_u2[sy][sx] - s_u1[sy][sx] + s_vp[sy][sx] * s_vp[sy][sx] * (du2_xx + du2_yy) * dt2dx2;

        // if ((gx == c_isrc-1) && (gy == c_jsrc-1))
        // {
        //     printf("\t%e %e %e %e %e %e\n", d_u3[idx], s_u2[sy][sx],s_u1[sy][sx],du2_xx, du2_yy,dt2dx2);
        //     printf("\t\t%i %i %f\n", ny, nx, dt2dx2);

        // }
        __syncthreads();

        // d_u3[idx] = tmp[sy+1][sx+1];
    }
}

/*
===================================================================================
MAIN
===================================================================================
*/
int main(int argc, char *argv[])
{
    // Model dimensions
    int nx = 512; /* x dim */
    int ny = 512; /* z dim */

    size_t nxy = nx * ny;
    size_t nbytes = nxy * sizeof(float); /* bytes to store nx * ny */

    float dx = 10.0; /* m */

    // Allocate memory for velocity model
    float _vp = 3300.0; /* m/s, p-wave velocity */
    float *h_vp;
    h_vp = (float *)malloc(nbytes);
    for (int i = 0; i < nxy; i++)
    {
        h_vp[i] = _vp;
    }

    printf("MODEL:\n");
    printf("\t%i x %i\t:ny x nx\n", ny, nx);
    printf("\t%f\t:dx\n", dx);
    printf("\t%f\t:h_vp[0]\n", h_vp[0]);

    // Time stepping
    float t_total = 0.55;         /* sec, total time of wave propagation */
    float dt = 0.35 * dx / _vp;    /* sec, time step assuming constant vp */
    int nt = round(t_total / dt); /* number of time steps */
    int snap_step = round(0.05 * nt);

    printf("TIME STEPPING:\n");
    printf("\t%e\t:t_total\n", t_total);
    printf("\t%e\t:dt\n", dt);
    printf("\t%i\t:nt\n", nt);

    // Source
    float f0 = 10.0;                 /* Hz, source dominant frequency */
    float t0 = 1.2 / f0;             /* source HALOding to move wavelet from left of zero */
    int isrc = round((float)nx / 2); /* source location, ox */
    int jsrc = round((float)ny / 2); /* source location, oz */

    float *h_wavelet, *h_time;
    float tbytes = nt * sizeof(float);
    h_time = (float *)malloc(tbytes);
    h_wavelet = (float *)malloc(tbytes);

    // Fill source waveform vecror
    float a = PI * PI * f0 * f0;        /* const for wavelet */
    float dt2dx2 = (dt * dt) / (dx * dx); /* const for fd stencil */
    for (size_t it = 0; it < nt; it++)
    {
        h_time[it] = it * dt;
        h_wavelet[it] = 1e10 * (1.0 - 2.0 * a * pow(h_time[it] - t0, 2)) * exp(-a * pow(h_time[it] - t0, 2));
        h_wavelet[it] *= dt2dx2;
    }

    printf("SOURCE:\n");
    printf("\t%f\t:f0\n", f0);
    printf("\t%f\t:t0\n", t0);
    printf("\t%i\t:isrc - ox\n", isrc);
    printf("\t%i\t:jsrc - oy\n", jsrc);
    printf("\t%e\t:dt2dx2\n", dt2dx2);
    printf("\t%f\t:min wavelength [m]\n",(float)_vp / (2*f0));
    printf("\t%f\t:ppw\n",(float)_vp / (2*f0) / dx);

    // Allocate memory on device
    printf("Allocate and copy memory on the device...\n");
    float *d_u1, *d_u2, *d_vp, *d_wavelet;
    CHECK(hipMalloc((void **)&d_u1, nbytes))       /* wavefield at t-2 */
    CHECK(hipMalloc((void **)&d_u2, nbytes))       /* wavefield at t-1 */
    CHECK(hipMalloc((void **)&d_vp, nbytes))       /* velocity model */
    CHECK(hipMalloc((void **)&d_wavelet, tbytes)); /* source term for each time step */

    CHECK(hipMemset(d_u1, 0, nbytes))
    CHECK(hipMemset(d_u2, 0, nbytes))
    CHECK(hipMemcpy(d_vp, h_vp, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_wavelet, h_wavelet, tbytes, hipMemcpyHostToDevice));

    float coef[] = {a0, a1, a2, a3, a4};
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_coef), coef, 5 * sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_isrc), &isrc, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_jsrc), &jsrc, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nx), &nx, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_ny), &ny, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nt), &nt, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_dt2dx2), &dt2dx2, sizeof(float)));
    printf("\t%f MB\n", (4 * nbytes + tbytes)/1024/1024);
    printf("OK\n");

    // Print out name of the main GPU
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, 0));
    printf("%s\t%d.%d:\n", deviceProp.name, deviceProp.major, deviceProp.minor);
    printf("%lu GB:\t total Global memory (gmem)\n", deviceProp.totalGlobalMem / 1024 / 1024 / 1000);
    printf("%lu MB:\t total Constant memory (cmem)\n", deviceProp.totalConstMem / 1024);
    printf("%lu MB:\t total Shared memory per block (smem)\n", deviceProp.sharedMemPerBlock / 1024);
    printf("%d:\t total threads per block\n", deviceProp.maxThreadsPerBlock);
    printf("%d:\t total registers per block\n", deviceProp.regsPerBlock);
    printf("%d:\t warp size\n", deviceProp.warpSize);
    printf("%d x %d x %d:\t max dims of block\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("%d x %d x %d:\t max dims of grid\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    CHECK(hipSetDevice(0));

    printf("CUDA:\n");
    printf("\t%i x %i\t:block dim\n", BDIMY, BDIMX);
    printf("\t%i x %i\t:shared dim\n", SDIMY, SDIMX);
    printf("CFL:\n");
    printf("\t%f\n", _vp * dt / dx);

    // Setup kernel run
    dim3 block(BDIMX, BDIMY);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    
    // MAIN LOOP
    printf("Time loop...\n");
    for (int it = 0; it < nt; it++)
    {
        printf("Step %i/%i\n", it+1, nt);
        // These kernels are in the same stream so they will be executes successively
        kernel_add_wavelet<<<grid, block>>>(d_u2, d_wavelet, it);
        kernel_2dfd<<<grid, block>>>(d_u1, d_u2, d_vp, dt2dx2);
        CHECK(hipDeviceSynchronize());

        // Exchange time steps
        float *d_u3 = d_u1;
        d_u1 = d_u2;
        d_u2 = d_u3;

        if ((it % snap_step == 0))
        {   
            saveSnapshotIstep(it, d_u3, nx, ny,"u3");
        }
    }
    printf("OK\n");

    CHECK(hipGetLastError());

    printf("Clean memory...");
    delete[] h_vp;
    delete[] h_time;
    delete[] h_wavelet;

    CHECK(hipFree(d_u1));
    CHECK(hipFree(d_u2));
    // CHECK(cudaFree(d_u3));
    CHECK(hipFree(d_vp));
    CHECK(hipFree(d_wavelet));
    printf("OK\n");

    CHECK(hipDeviceReset());

    return 0;
}
